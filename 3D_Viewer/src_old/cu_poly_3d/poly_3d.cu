#include "hip/hip_runtime.h"
/*
*  file:  poly_3d.cu
*
*  Host and device code for CUDA poly_3d.
*
*  $Id: //depot/gsg/HHMI/Phase2/src/cu_poly_3d/poly_3d.cu#14 $
*  $Date: 2009/11/30 $
*  $Author: rkneusel $
*
*  RTK, 24-Feb-2009
*  Last update:  16-Nov-2009
*/

#include <pthread.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <fcntl.h>
#ifndef WIN32
#include <unistd.h>
#include <sys/time.h>
#endif

#include "poly_3d.h"

//#ifndef WIN32
//#define DEBUG
//#endif

//  Thread stuff
typedef struct thread_data {
    int start;
    int end;
    int nx;
    int ny;
    unsigned short *img;
    unsigned short *out;
    float *P;
    float *Q;
} thread_data_t;

//  x,y to offset
__device__ int stride;
#define IDX(X,Y)  ((X)+(Y)*(stride))

//  Number of threads per block
#define THREADS     256

//  Coefficient arrays
__device__ __constant__ float d_p[MAX_FRAMES*2*2];
__device__ __constant__ float d_q[MAX_FRAMES*2*2];

unsigned short *d_out;  //  Output image stack 
unsigned short *d_img;  //  Input image stack
unsigned short *d_tmp;  //  Single output image

//
//  Device code:
//


/**************************************************************
*  bilinear
*
*  Use bilinear interpolation to get the output image value.
*
*  img     -  source image
*  a, b    -  coordinates to interpolate at
*  nx, ny  -  image dimensions
*/
__device__ unsigned short bilinear(unsigned short *img, float a, float b, int nx, int ny) {
  int x1,x2,y1,y2;
  unsigned short q11, q12, q21, q22;
  float A,B,C,D;
  unsigned short ans;

  //  Set the stride for IDX
  stride = nx;
  
  //  Get the coords around (a,b)
  x1 = (int)floor(a);
  y1 = (int)floor(b);
  
  //  Check for out of bounds
  if ((x1 < 0) || (y1 < 0) || (x1 > nx-2) || (y1 > ny-2)) {
    x1 = (x1 < 0) ? 0 : x1;
    x1 = (x1 > nx-1) ? nx-1 : x1;
    y1 = (y1 < 0) ? 0 : y1;
    y1 = (y1 > ny-1) ? ny-1 : y1;
    ans = img[IDX(x1,y1)];
  } else {
    x2 = x1 + 1;
    y2 = y1 + 1;
    
    //  Get the image values at the coordinates above
    q11 = img[IDX(x1,y1)];
    q12 = img[IDX(x1,y2)];
    q21 = img[IDX(x2,y1)];
    q22 = img[IDX(x2,y2)];
    
    A = x2-a;
    B = y2-b;
    C = a-x1;
    D = b-y1;
    
    ans = (unsigned short)(0.5+(A*B*q11 + C*B*q21 + A*D*q12 + C*D*q22));
  }
  __syncthreads();

  return ans;
}


/**************************************************************
*  poly2d
*
*  Interpolate a single image.
*
*  img     -  input image (nx,ny)
*  out     -  output image pointer
*  nx, ny  -  image dimensions
*  mp      -  index into d_q and d_p
*/
__global__ void poly2d(unsigned short *img, unsigned short *out, int nx, int ny, int mp) {
  int x,y;
  float a,b;
  
  //  index into the image
  int tid = threadIdx.x + blockIdx.x*blockDim.x;

  //  Convert tid to x,y (col,row) indices
  x = tid / nx;
  y = tid - nx*x;

  //  Calculate a(x,y) and b(x,y), indices into original image
  a = d_p[mp+0] + d_p[mp+1]*x + d_p[mp+2]*y + d_p[mp+3]*x*y;
  b = d_q[mp+0] + d_q[mp+1]*x + d_q[mp+2]*y + d_q[mp+3]*x*y;

  //  Calculate interpolated image value
  out[tid] = bilinear(img, a, b, nx, ny);
}


//
//  Host code:
//

#ifdef DEBUG
/**************************************************************
*  getTime
*
*  Return the system time.
*/
double getTime() {
    struct timeval tv;
    hipDeviceSynchronize();
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
}
#endif

/**************************************************************
*  cuda_error
*
*  Check for a CUDA error on the last call.  If there is one
*  return the code (else hipSuccess) and set the pointer, if
*  not NULL, to the message text.
*/
void checkError(char *s) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr, "ERROR: %s: %s\n", s, hipGetErrorString(err));
  }
}


/**************************************************************
*  cuda_safe_init
*
*  Wait until all threads done.
*/
extern "C" void cuda_safe_init(void) {
  hipDeviceSynchronize();
}


/**************************************************************
*  poly_warp_3d
*
*  Thread entry point
*/
void *poly_warp_3d(void *arg) {
    thread_data_t *data = (thread_data_t *)arg;
    int nframes, nblocks, nx, ny, nelem;
    int ip, mp, k;
    unsigned short *img, *out;
#ifdef DEBUG
    double s,e;
#endif

    //  Input and output pointers
    img = data->img;
    out = data->out;
    
    //  Set the card to use
    hipSetDevice((data->start != 0));

    //  Copy P and Q to the device
    nframes = data->end - data->start + 1;

    hipMemcpyToSymbol(HIP_SYMBOL(d_p), &(data->P)[data->start], nframes*2*2*sizeof(float));
    checkError("Copy P to d_p");
    hipDeviceSynchronize();

    hipMemcpyToSymbol(HIP_SYMBOL(d_q), &(data->Q)[data->start], nframes*2*2*sizeof(float));
    checkError("Copy Q to d_q");
    hipDeviceSynchronize();

    //  Allocate memory for the image stack
    nx = data->nx;
    ny = data->ny;
    nelem = nx*ny*nframes;
    hipMalloc((void **)&d_img, nelem*sizeof(unsigned short));
    checkError("Allocate d_img");
    hipDeviceSynchronize();

    //  Allocate memory for the output image stack
    hipMalloc((void **)&d_out, nelem*sizeof(unsigned short));
    checkError("Allocate d_out");
    hipDeviceSynchronize();

    //  Temporary output image
    nblocks = (int)ceil((nx*ny)/(float)THREADS);
    hipMalloc((void **)&d_tmp, nblocks*THREADS*sizeof(unsigned short));
    checkError("Allocate d_tmp");
    hipDeviceSynchronize();

    //  Copy the input images to the device
    hipMemcpy(d_img, &img[data->start], nelem*sizeof(unsigned short), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    //  Set up the grid and blocks for one image
    dim3 threads(THREADS);
    dim3 blocks(nblocks);

    //
    //  Loop over all images in the stack
    //
#ifdef DEBUG
    s = getTime();
#endif
    for(k=0; k < nframes; k++) {
        //  Interpolate the current frame
        ip = nx*ny*k;  // current image index
        mp = 2*2*k;    // current P,Q matrix index
        poly2d<<< blocks, threads >>>(&d_img[ip], d_tmp, nx, ny, mp);
        checkError("poly2d call");
        hipDeviceSynchronize();
        
        //  Copy the new image to the output image stack
        hipMemcpy(&d_out[ip], d_tmp, nx*ny*sizeof(unsigned short),
                   hipMemcpyDeviceToDevice);
        checkError("Device to device memcpy");
        hipDeviceSynchronize();
    }
#ifdef DEBUG
    e = getTime();
    printf("Kernel call = %f\n", e-s);
#endif

  //  Copy the output image directly to IDL memory
#ifdef DEBUG
    s = getTime();
#endif
    hipMemcpy(&out[data->start], d_out, nelem*sizeof(unsigned short), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
#ifdef DEBUG
    e = getTime();
    printf("Copy image stack from device = %f\n", e-s);
#endif

    //  Clean up device memory
    hipFree(d_img);
    checkError("d_img free");
    hipFree(d_out);
    checkError("d_out free");
    hipFree(d_tmp);
    checkError("d_tmp free");

    return 0;
}


/**************************************************************
*  poly_3d
*
*  Entry point from IDL DLM code.
*
*  img, nx, ny, nz  -  input image and dimensions
*  P, Q             -  polynomial coeff, 2x2xnz assumed
*  out              -  output image, same dims as img
*/
extern "C" void poly_3d(unsigned short *img, int nx, int ny, int nframes,
                        float *P, float *Q,
                        unsigned short *out) {
  pthread_t thing1, thing2;
  pthread_attr_t attr;
  thread_data_t data1, data2;
  void *status;
  int rc, ncards;
#ifdef DEBUG
  double s,e;
#endif

#ifdef DEBUG
  s = getTime();
#endif

  //  How many cards in the system?
  hipGetDeviceCount(&ncards);

  //  Ensure threads are joinable
  pthread_attr_init(&attr);
  pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);

  //  Set up pointers to divide the input image stack into two parts
  data1.start = 0;
  data2.start = nframes / 2;
  data1.end = data2.start - 1;
  data2.end = nframes - 1;
  data1.nx = data2.nx = nx;
  data1.ny = data2.ny = ny;
  data1.img = data2.img = img;
  data1.out = data2.out = out;
  data1.P = data2.P = P;
  data1.Q = data2.Q = Q;

  //  Adjust if only one card
  if (ncards == 1) {
    data1.end = nframes - 1;
  }

  //  Create the threads, each one processing one part of the image stack
  if ((rc=pthread_create(&thing1, &attr, poly_warp_3d, (void *)&data1))) {
    printf("Error: Unable to create thread: %d\n", rc);
  }
  
  if (ncards > 1) {
    if ((rc=pthread_create(&thing2, &attr, poly_warp_3d, (void *)&data2))) {
      printf("Error: Unable to create thread: %d\n", rc);
    }
  }

  //  Wait for the threads to finish
  pthread_attr_destroy(&attr);
  pthread_join(thing1, &status);
  
  if (ncards > 1) {
    pthread_join(thing2, &status);
  }

#ifdef DEBUG
  e = getTime();
  printf("Run time = %f\n", e-s);
#endif
}

#ifdef DEBUG
/**************************************************************
*  main
*
*  A simple driver for testing the kernel.  Build with:
*
*  $ nvcc poly_3d.cu -o poly_3d
*
*/
int main(int argc, char *argv[]) {
  unsigned short *img, *out;
  float *p, *q;
  double s,e;

  img = (unsigned short *)malloc(512*512*100*sizeof(unsigned short));
  out = (unsigned short *)malloc(512*512*100*sizeof(unsigned short));
  p = (float *)malloc(2*2*100*sizeof(float));
  q = (float *)malloc(2*2*100*sizeof(float));

  s = getTime();
  poly_3d(img, 512, 512, 100, p, q, out);
  e = getTime();
  printf("Run time = %f\n", e-s);

  return 0;
}
#endif

/*
*  end poly_3d.cu
*/

