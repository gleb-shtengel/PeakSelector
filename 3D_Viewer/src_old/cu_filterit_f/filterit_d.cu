#include "hip/hip_runtime.h"
/*
*  file:  filterit_d.cu
*
*  Double precision host and device code for CUDA filter.
*
*  $Id: //depot/gsg/HHMI/Phase2/src/cu_filter/filter_f.cu#1 $
*  $Date: 2009/03/10 $
*  $Author: rkneusel $
*
*  RTK, 25-Jun-2009
*  Last update:  28-Jul-2009
*/

#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#ifndef WIN32
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/time.h>
#endif

//  Number of threads per block
#define THREADS     256

//  Parameter limits
#define MAX_LIMITS    5
#define MAX_PARAMS 1000
#define MAX_PARAM_SPACE  (MAX_PARAMS*MAX_LIMITS)

__device__ __constant__ float d_params[MAX_PARAM_SPACE];
__device__ __constant__ unsigned char d_index[MAX_PARAMS];

double *d_gp;           //  CGroupParams
unsigned char *d_out;  //  Output filter flags

//
//  Device code:
//

/**************************************************************
*  check
*
*  Check a particular element of gp
*
*  row     -  Data values for the current CGroupParams peak
*  col     -  Column number to check
*  nparams -  Number of parameters
*/
__device__ unsigned char check(double *row, int col, int nparams) {
  double d = row[col];
  
  return ((d >= d_params[col]) && (d <= d_params[col+nparams]));
}


/**************************************************************
*  k_filter
*
*  FilterIt kernel
*
*  gp      -  CGroupParams on device
*  out     -  Filter output vector
*  nparams -  Number of parameters
*/
__global__ void k_filter(double *gp, unsigned char *out, int nparams) {
  int tid, i;
  double *row;
  unsigned char ans = 1;
  
  //  Output flag index
  tid = threadIdx.x + blockIdx.x*blockDim.x;

  //  Point to the proper row
  row = &gp[nparams*tid];

  //  Check
  for(i=0; i < nparams; i++) {
    if (d_index[i])
      ans &= check(row, i, nparams);
    __syncthreads();
  }
  
  //  Assign the output value
  out[tid] = ans;
}


//
//  Host code:
//

#ifndef WIN32
/**************************************************************
*  getTime
*
*  Return the system time.
*/
double getTime() {
    struct timeval tv;
    hipDeviceSynchronize();
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
}
#endif


/**************************************************************
*  checkError
*
*  Check for a CUDA error on the last call.
*/
void checkError(char *s) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr, "ERROR: %s: %s\n", s, hipGetErrorString(err));
  }
}


/**************************************************************
*  cudaThreadSync
*/
void cudaThreadSync() {
  hipError_t err = hipDeviceSynchronize();

  if (hipSuccess != err) {
    fprintf(stderr, "ERROR: Bad sync: %s\n", hipGetErrorString(err));
  }
}


/**************************************************************
*  cuda_safe_init
*
*  Wait until all threads done.
*/
extern "C" void cuda_safe_init(void) {
  hipDeviceSynchronize();
}


/**************************************************************
*  filterit
*
*  Entry point from IDL DLM code.
*
*/
extern "C" void filterit(double *gp, int nparams, int npeaks, int nlimits, float *params,
                         unsigned char *filterindex, unsigned char *out) {
  int nblocks;

  //  Copy ParamLimits to the device
  hipMemcpyToSymbol(HIP_SYMBOL(d_params), params, nparams*nlimits*sizeof(float));
  checkError("copy ParamLimits");
  cudaThreadSync();

  //  Copy filterindex to the device
  hipMemcpyToSymbol(HIP_SYMBOL(d_index), filterindex, nparams*sizeof(unsigned char));
  checkError("copy filterindex");
  cudaThreadSync();

  //  Allocate memory for the output filter flags
  nblocks = (int)ceil(npeaks/(float)THREADS);
  hipMalloc((void **)&d_out, nblocks*THREADS*sizeof(unsigned char));
  checkError("allocate d_out");
  cudaThreadSync();

  //  Allocate memory for CGroupParams aligned properly
  hipMalloc((void **)&d_gp, nparams*nblocks*THREADS*sizeof(double));
  checkError("allocate CGroupParams");
  cudaThreadSync();

  //  Copy CGroupParams to the device
  hipMemcpy(d_gp, gp, nparams*npeaks*sizeof(double), hipMemcpyHostToDevice);
  checkError("copy CGroupParams");
  cudaThreadSync();

  //  Set up the grid and blocks for one image
  dim3 threads(THREADS);
  dim3 blocks(nblocks);

  //
  //  Call the kernel
  //
  k_filter<<< nblocks, THREADS >>>(d_gp, d_out, nparams);
  checkError("kernel call");
  cudaThreadSync();

  //  Copy the output directly to IDL memory
  hipMemcpy(out, d_out, npeaks*sizeof(unsigned char), hipMemcpyDeviceToHost);
  checkError("copy output");
  cudaThreadSync();

  //  Clean up device memory
  hipFree(d_gp);
  checkError("d_gp free");
  hipFree(d_out);
  checkError("d_out free");
}


#ifndef WIN32
/**************************************************************
*  main
*
*  A simple driver for testing the kernel.  Build with:
*
*  $ nvcc filterit_d.cu -o filterit_d
*
*/
int main(int argc, char *argv[]) {
  double *gp;
  float *p;
  unsigned char *out, *index;
  double s,e;
  
  p = (float *)malloc(45*5*sizeof(double));
  gp = (double *)malloc(45*148623*sizeof(double));
  out = (unsigned char *)malloc(148623);
  index = (unsigned char *)malloc(45);

  memset((void *)index, 0, 45*sizeof(unsigned char));
  memset((void *)index, 1, 10*sizeof(unsigned char)); 

  s = getTime();
  filterit(gp, 45, 148623, 5, p, index, out);
  e = getTime();
  printf("Total runtime = %f\n", e-s);

  free(p);
  free(gp);
  free(out);
  return 0;
}
#endif

/*
*  end filterit_d.cu
*/

